#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "header.h"


#define CHECK_ERR(err,msg) (\
		{if (err != hipSuccess) { \
			fprintf(stderr, msg " - %s\n", hipGetErrorString(err)); \
			exit(EXIT_FAILURE); \
		} \
	})

//////////////////////////////////////////////////////////////////////////////////////////////////calculate mutant score

__device__ float calcMutantScore(char* seq1, char* seq2, float* weights, float* lettersGrid, int len2, int n, int k)
{
	float score = 0;
	int i = 0, j = 0;
	for (i = 0; i < len2; i++, j++)
	{
		if (j == n || j == k) 
			j++;
		int fc_idx = seq1[i] - 65;
		int sc_idx = seq2[j] - 65;
		score += lettersGrid[fc_idx*ABC_NUM + sc_idx];
	}	

	return score;	
}

//////////////////////////////////////////////////////////////////////////////////////////////// find mutant best score

__global__ void getMutantBestScore(char* d_seq1, char* d_seq2, float* d_weights, 
						float* d_bestScores, int* d_bestOffsets, int* d_nkArr, float* d_lettersGrid,
						int num_mutants, int maxOffset, int len2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float bestScore = -10000;
	int offset = 0;
	
	if (i < num_mutants)
	{
		for (int j = 0; j < maxOffset; j++)
		{
			float score = calcMutantScore(d_seq1 + j,d_seq2, d_weights, d_lettersGrid, len2, d_nkArr[i], d_nkArr[i+num_mutants]);
			if (score > bestScore)
			{
				bestScore = score;
				offset = j;
			}
		}
		d_bestScores[i] = bestScore;
		d_bestOffsets[i] = offset;
	}		
} 

void calcBestScoreCUDA(char* seq1, char* seq2, float weights[], float* bestScores, int* bestOffsets, int num_mutants, int len2, int* nkArr, float* lettersGrid)
{
	int len1 = strlen(seq1);
	int maxOffset = len1 - (len2-2) + 1;

////////////////////////////////////////////////////////////////////////////////////////////// allocate seq1 memory

	char* d_seq1 = NULL; 
	
	hipError_t err = hipSuccess;
	size_t  arrSize = len1 * sizeof(char);
	err = hipMalloc((void**)&d_seq1, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	err = hipMemcpy(d_seq1, seq1, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device"); 
	
////////////////////////////////////////////////////////////////////////////////////////////// allocate seq2 memory	

	char* d_seq2 = NULL; 
	
	err = hipSuccess;
	arrSize = len2 * sizeof(char);
	hipMalloc((void**)&d_seq2, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_seq2, seq2, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device"); 

////////////////////////////////////////////////////////////////////////////////////////////// allocate weights memory

	float* d_weights = NULL; 
	
	err = hipSuccess;
	arrSize = WEIGHTS_NUM * sizeof(float);
	hipMalloc((void**)&d_weights, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_weights, weights, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device");
	
////////////////////////////////////////////////////////////////////////////////////////////// allocate weights memory

	int* d_nkArr = NULL; 

	err = hipSuccess;
	arrSize = num_mutants*2 * sizeof(int);
	hipMalloc((void**)&d_nkArr, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_nkArr, nkArr, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device"); 

////////////////////////////////////////////////////////////////////////////////////////////// allocate weights memory

	float* d_lettersGrid = NULL; 
	
	err = hipSuccess;
	arrSize = ABC_NUM*ABC_NUM * sizeof(float);
	hipMalloc((void**)&d_lettersGrid, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_lettersGrid, lettersGrid, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device"); 

//////////////////////////////////////////////////////////////////////////////////////////// allocate best scores memory

	float* d_bestScores = NULL;  
	
	err = hipSuccess;
	arrSize = num_mutants * sizeof(float);
	hipMalloc((void**)&d_bestScores, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_bestScores, bestScores, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device"); 

//////////////////////////////////////////////////////////////////////////////////////////// allocate best offsets memory

	int* d_bestOffsets = NULL;
	
	err = hipSuccess;
	arrSize = num_mutants * sizeof(int);
	hipMalloc((void**)&d_bestOffsets, arrSize);
	CHECK_ERR(err, "Failed to allocate device memory");
	hipMemcpy(d_bestOffsets, bestOffsets, arrSize, hipMemcpyHostToDevice);
	CHECK_ERR(err, "Failed to copy data from host to device");
	
	int threads = 256;
	int blocks = (num_mutants + threads-1) / threads;
	
	getMutantBestScore<<<blocks, threads>>>(d_seq1, d_seq2, d_weights, d_bestScores, d_bestOffsets, d_nkArr, d_lettersGrid, num_mutants, maxOffset, len2);
	
	hipMemcpy(bestScores, d_bestScores, num_mutants * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(bestOffsets, d_bestOffsets, num_mutants * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(d_seq1);
	hipFree(d_seq2);
	hipFree(d_weights);
	hipFree(d_nkArr);
	hipFree(d_bestScores);
	hipFree(d_bestOffsets);
}

// ========================

